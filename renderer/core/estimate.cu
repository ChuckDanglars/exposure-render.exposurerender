
#include "estimate.cuh"
#include "core\cudawrapper.h"
#include "core\renderer.h"

namespace ExposureRender
{

KERNEL void KrnlEstimate(Renderer* Renderer)
{
	const int X 	= blockIdx.x * blockDim.x + threadIdx.x;
	const int Y		= blockIdx.y * blockDim.y + threadIdx.y;

	if (X >= Renderer->Camera.GetFilm().GetWidth() || Y >= Renderer->Camera.GetFilm().GetHeight())
		return;
	
	CudaBuffer2D<ColorXYZAf>& IterationEstimateHDR = Renderer->Camera.GetFilm().GetIterationEstimateHDR();

	RNG Random = Renderer->Camera.GetFilm().GetRandomNumberGenerator(Vec2i(X, Y));

	Ray R;

	Renderer->Camera.Sample(R, Vec2i(X, Y), Random);

	ScatterEvent SE;

	if (Renderer->Volume.Intersect(R, Random, SE))
		IterationEstimateHDR.Set(X, Y, ColorXYZAf(1.0f, 1.0f, 1.0f, 0.0f));
	else
		IterationEstimateHDR.Set(X, Y, ColorXYZAf(0.0f, 0.0f, 0.0f, 0.0f));
	/*

	

	float T[2] = { 0.0f };

	bool Intersects = B.Intersect(R, T[0], T[1]);

	Renderer->Camera.GetFilm().GetIterationEstimateHDR().Set(X, Y, ColorXYZAf(Intersects ? 1.0f : 0.0f, 0.0f, 0.0f, 0.0f));

	
	Output[PID * 3 + 0] = Intersects ? 255 : 0;
	Output[PID * 3 + 1] = 0;
	Output[PID * 3 + 2] = 0;
	*/
}

void Estimate(Renderer* HostRenderer, Renderer* DevRenderer)
{
	LAUNCH_DIMENSIONS

	KrnlEstimate<<<Grid, Block>>>(DevRenderer);
	hipDeviceSynchronize();
	Cuda::HandleCudaError(hipGetLastError(), "Estimate");
}

}
